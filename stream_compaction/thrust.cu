#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());


            thrust::host_vector<int> host_idata(idata, idata + n);

            thrust::device_vector<int> dev_idata(host_idata);
            thrust::device_vector<int> dev_odata(n);
            hipDeviceSynchronize();



            thrust::device_vector<int> d_in(host_idata); 
            thrust::device_vector<int> d_out(n);
            hipDeviceSynchronize();

            timer().startGpuTimer();
            thrust::exclusive_scan(d_in.begin(), d_in.end(), d_out.begin());
            timer().endGpuTimer();

            thrust::copy(d_out.begin(), d_out.end(), odata);

            hipDeviceSynchronize();



            
        }
    }
}
